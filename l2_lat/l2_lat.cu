//This code is a modification of L1 cache benchmark from 
//"Dissecting the NVIDIA Volta GPU Architecture via Microbenchmarking": https://arxiv.org/pdf/1804.06826.pdf

//This benchmark measures the latency of L2 latency using pointer-chasing

//This code have been tested on Volta V100 architecture

#include <stdio.h>   
#include <stdlib.h> 
#include <hip/hip_runtime.h>

#define THREADS_NUM 1     // one thread to initialize the pointer-chasing array
#define WARP_SIZE 32
#define ITERS 32768        //iterate over the array ITERS times
#define ARRAY_SIZE 4096

// GPU error check
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true){
        if (code != hipSuccess) {
                fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
                if (abort) exit(code);
        }
}

__global__ void l2_lat(uint32_t *startClk, uint32_t *stopClk, uint64_t *posArray, uint64_t *dsink){

	// thread index
	uint32_t tid = threadIdx.x;

	// initialize pointer-chasing array with just one thread
	if (tid == 0){
		for (uint32_t i=0; i<(ARRAY_SIZE-1); i++)
			posArray[i] = (uint64_t)(posArray + i + 1);

		posArray[ARRAY_SIZE-1] = (uint64_t)posArray;
	}

	if(tid < THREADS_NUM){

		uint64_t *ptr = posArray + tid;
		uint64_t ptr1, ptr0;	

		// initialize the pointers with the start address
		// use cg modifier to cache the load in L2 and bypass L1
		asm volatile ("{\t\n"
			"ld.global.cg.u64 %0, [%1];\n\t"
			"}" : "=l"(ptr1) : "l"(ptr) : "memory"
		);
	
		// synchronize all threads
		asm volatile ("bar.sync 0;");

		// start timing
		uint32_t start = 0;
		asm volatile ("mov.u32 %0, %%clock;" : "=r"(start) :: "memory");

		// pointer-chasing ITERS times
		// use cg modifier to cache the load in L2 and bypass L1
		for(uint32_t i=0; i<ITERS; ++i) {	
			asm volatile ("{\t\n"
				"ld.global.cg.u64 %0, [%1];\n\t"
				"}" : "=l"(ptr0) : "l"((uint64_t*)ptr1) : "memory"
			);
			ptr1 = ptr0;    //swap the register for the next load
		}

		// stop timing
		uint32_t stop = 0;
		asm volatile("mov.u32 %0, %%clock;" : "=r"(stop) :: "memory");

		// write time and data back to memory
		startClk[tid] = start;
		stopClk[tid] = stop;
		dsink[tid] = ptr1;
	}
}

int main(){
	uint32_t *startClk = (uint32_t*) malloc(THREADS_NUM*sizeof(uint32_t));
	uint32_t *stopClk = (uint32_t*) malloc(THREADS_NUM*sizeof(uint32_t));
	uint64_t *dsink = (uint64_t*) malloc(THREADS_NUM*sizeof(uint64_t));
	
	uint32_t *startClk_g;
        uint32_t *stopClk_g;
        uint64_t *posArray_g;
        uint64_t *dsink_g;
	
	gpuErrchk( hipMalloc(&startClk_g, THREADS_NUM*sizeof(uint32_t)) );
	gpuErrchk( hipMalloc(&stopClk_g, THREADS_NUM*sizeof(uint32_t)) );
	gpuErrchk( hipMalloc(&posArray_g, ARRAY_SIZE*sizeof(uint64_t)) );
	gpuErrchk( hipMalloc(&dsink_g, THREADS_NUM*sizeof(uint64_t)) );
	
	l2_lat<<<1,THREADS_NUM>>>(startClk_g, stopClk_g, posArray_g, dsink_g);
	gpuErrchk( hipPeekAtLastError() );

	gpuErrchk( hipMemcpy(startClk, startClk_g, THREADS_NUM*sizeof(uint32_t), hipMemcpyDeviceToHost) );
	gpuErrchk( hipMemcpy(stopClk, stopClk_g, THREADS_NUM*sizeof(uint32_t), hipMemcpyDeviceToHost) );
	gpuErrchk( hipMemcpy(dsink, dsink_g, THREADS_NUM*sizeof(uint64_t), hipMemcpyDeviceToHost) );
	printf("L1 Latency for %d threads = %u \n", THREADS_NUM, (stopClk[0]-startClk[0]));

	return 0;
} 
