//This code is a modification of L2 cache benchmark from 
//"Dissecting the NVIDIA Volta GPU Architecture via Microbenchmarking": https://arxiv.org/pdf/1804.06826.pdf

//This benchmark measures the read latency of L2 cache for 64f

//This code have been tested on Volta V100 architecture

#include <stdio.h>   
#include <stdlib.h> 
#include <hip/hip_runtime.h>

#define TEST_SIZE 1
#define WARP_SIZE 32
#define L1_SIZE 16384
#define THREADS_NUM 1024
#define ARRAY_SIZE (TEST_SIZE+L1_SIZE)

// GPU error check
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true){
        if (code != hipSuccess) {
                fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
                if (abort) exit(code);
        }
}

__global__ void l2_lat(uint32_t *startClk, uint32_t *stopClk, double *dsink, double *posArray){
	
	// thread index
	uint32_t tid = threadIdx.x;
	if(tid < TEST_SIZE){
	// a register to avoid compiler optimization
	double sink = 0;
	// populate l2 cache to warm up
	if (tid<TEST_SIZE){
		double *ptr = posArray + tid + L1_SIZE;
		asm volatile ("{\t\n"
			".reg .f64 data;\n\t"
			"ld.global.cg.f64 data, [%1];\n\t"
			"add.f64 %0, data, %0;\n\t"
			"}" : "+d"(sink) : "l"(ptr) : "memory"
		);
	}

	// synchronize all threads
	asm volatile ("bar.sync 0;");

	for (uint32_t i=tid; i<L1_SIZE; i+=THREADS_NUM){
                double *ptr = posArray + i;
                asm volatile ("{\t\n"
                        ".reg .f64 data;\n\t"
                        "ld.global.cg.f64 data, [%1];\n\t"
                        "add.f64 %0, data, %0;\n\t"
                        "}" : "+d"(sink) : "l"(ptr) : "memory"
                );
	}
	
        // synchronize all threads
        asm volatile ("bar.sync 0;");

	// start timing
	uint32_t start = 0;
	asm volatile ("mov.u32 %0, %%clock;" : "=r"(start) :: "memory");
	
	// load data from l2 cache and accumulate
	
        if (tid<TEST_SIZE){
                double *ptr = posArray + tid + L1_SIZE;
                asm volatile ("{\t\n"
                        ".reg .f64 data;\n\t"
                        "ld.global.cg.f64 data, [%1];\n\t"
                        "add.f64 %0, data, %0;\n\t"
                        "}" : "+d"(sink) : "l"(ptr) : "memory"
                );
        }

	// synchronize all threads
	asm volatile("bar.sync 0;");
	
	// stop timing
	uint32_t stop = 0;
	asm volatile("mov.u32 %0, %%clock;" : "=r"(stop) :: "memory");
	// write time and data back to memory
	startClk[tid] = start;
	stopClk[tid] = stop;
	dsink[tid] = sink;
	}
}

int main(){
	uint32_t *startClk = (uint32_t*) malloc(THREADS_NUM*sizeof(uint32_t));
	uint32_t *stopClk = (uint32_t*) malloc(THREADS_NUM*sizeof(uint32_t));
	double *posArray = (double*) malloc(ARRAY_SIZE*sizeof(double));
	double *dsink = (double*) malloc(THREADS_NUM*sizeof(double));
	
	uint32_t *startClk_g;
        uint32_t *stopClk_g;
        double *posArray_g;
        double *dsink_g;
	
	for (uint32_t i=0; i<ARRAY_SIZE; i++)
		posArray[i] = (double)i;
		
	gpuErrchk( hipMalloc(&startClk_g, THREADS_NUM*sizeof(uint32_t)) );
	gpuErrchk( hipMalloc(&stopClk_g, THREADS_NUM*sizeof(uint32_t)) );
	gpuErrchk( hipMalloc(&posArray_g, ARRAY_SIZE*sizeof(double)) );
	gpuErrchk( hipMalloc(&dsink_g, THREADS_NUM*sizeof(double)) );
	
	gpuErrchk( hipMemcpy(posArray_g, posArray, ARRAY_SIZE*sizeof(double), hipMemcpyHostToDevice) );

	l2_lat<<<1,THREADS_NUM>>>(startClk_g, stopClk_g, dsink_g, posArray_g);

	gpuErrchk( hipMemcpy(startClk, startClk_g, THREADS_NUM*sizeof(uint32_t), hipMemcpyDeviceToHost) );
	gpuErrchk( hipMemcpy(stopClk, stopClk_g, THREADS_NUM*sizeof(uint32_t), hipMemcpyDeviceToHost) );
	gpuErrchk( hipMemcpy(dsink, dsink_g, THREADS_NUM*sizeof(double), hipMemcpyDeviceToHost) );
	printf("L2 Latency for %d threads = %u \n", TEST_SIZE, stopClk[0]-startClk[0]);

	return 0;
} 
