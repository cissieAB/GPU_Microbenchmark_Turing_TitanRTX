//This code is a modification of L1 cache benchmark from 
//"Dissecting the NVIDIA Volta GPU Architecture via Microbenchmarking": https://arxiv.org/pdf/1804.06826.pdf

//This benchmark measures the maximum read bandwidth of L1 cache

//This code have been tested on Volta V100 architecture

#include <stdio.h>   
#include <stdlib.h> 
#include <hip/hip_runtime.h>

#define THREADS_NUM 32
#define WARP_SIZE 32
#define L1_SIZE 32768

// GPU error check
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true){
        if (code != hipSuccess) {
                fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
                if (abort) exit(code);
        }
}

__global__ void l1_bw(uint32_t *startClk, uint32_t *stopClk, float *dsink, float *posArray){
	
	// thread index
	uint32_t tid = threadIdx.x;
	if(tid < THREADS_NUM){
	// a register to avoid compiler optimization
	float sink = 0;
	float *ptr = posArray + tid;
	// populate l1 cache to warm up
	asm volatile ("{\t\n"
		".reg .f32 data;\n\t"
		"ld.global.ca.f32 data, [%1];\n\t"
		"add.f32 %0, data, %0;\n\t"
		"}" : "+f"(sink) : "l"(ptr) : "memory"
	);
	
	// synchronize all threads
	asm volatile ("bar.sync 0;");
	
	// start timing
	uint32_t start = 0;
	asm volatile ("mov.u32 %0, %%clock;" : "=r"(start) :: "memory");
	
	// load data from l1 cache and accumulate
	asm volatile ("{\t\n"
		".reg .f32 data;\n\t"
		"ld.global.ca.f32 data, [%1];\n\t"
		"add.f32 %0, data, %0;\n\t"
		"}" : "+f"(sink) : "l"(ptr) : "memory"
	);

	// synchronize all threads
	asm volatile("bar.sync 0;");
	
	// stop timing
	uint32_t stop = 0;
	asm volatile("mov.u32 %0, %%clock;" : "=r"(stop) :: "memory");
	// write time and data back to memory
	startClk[tid] = start;
	stopClk[tid] = stop;
	dsink[tid] = sink;
	}
}

int main(){
	uint32_t *startClk = (uint32_t*) malloc(THREADS_NUM*sizeof(uint32_t));
	uint32_t *stopClk = (uint32_t*) malloc(THREADS_NUM*sizeof(uint32_t));
	float *posArray = (float*) malloc(THREADS_NUM*sizeof(float));
	float *dsink = (float*) malloc(THREADS_NUM*sizeof(float));
	
	uint32_t *startClk_g;
        uint32_t *stopClk_g;
        float *posArray_g;
        float *dsink_g;
	
	for (uint32_t i=0; i<THREADS_NUM; i++)
		posArray[i] = (float)i;
		
	gpuErrchk( hipMalloc(&startClk_g, THREADS_NUM*sizeof(uint32_t)) );
	gpuErrchk( hipMalloc(&stopClk_g, THREADS_NUM*sizeof(uint32_t)) );
	gpuErrchk( hipMalloc(&posArray_g, THREADS_NUM*sizeof(float)) );
	gpuErrchk( hipMalloc(&dsink_g, THREADS_NUM*sizeof(float)) );
	
	gpuErrchk( hipMemcpy(posArray_g, posArray, THREADS_NUM*sizeof(float), hipMemcpyHostToDevice) );
	l1_bw<<<1,THREADS_NUM>>>(startClk_g, stopClk_g, dsink_g, posArray_g);
	gpuErrchk( hipMemcpy(startClk, startClk_g, THREADS_NUM*sizeof(uint32_t), hipMemcpyDeviceToHost) );
	gpuErrchk( hipMemcpy(stopClk, stopClk_g, THREADS_NUM*sizeof(uint32_t), hipMemcpyDeviceToHost) );
	gpuErrchk( hipMemcpy(dsink, dsink_g, THREADS_NUM*sizeof(float), hipMemcpyDeviceToHost) );
	printf("L1 Latency for %d threads = %u \n", THREADS_NUM, stopClk[0]-startClk[0]);

	return 0;
} 
