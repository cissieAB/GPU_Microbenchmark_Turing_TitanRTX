//This code is a modification of L1 cache benchmark from 
//"Dissecting the NVIDIA Volta GPU Architecture via Microbenchmarking": https://arxiv.org/pdf/1804.06826.pdf

//This benchmark measures the latency of L1 cache 32f read

//This code have been tested on Volta V100 architecture

#include <stdio.h>   
#include <stdlib.h> 
#include <hip/hip_runtime.h>

#define BLOCKS_NUM 80
#define THREADS_NUM 32
#define TOTAL_THREADS (BLOCKS_NUM*THREADS_NUM)
#define WARP_SIZE 32
#define ITERS 32768
#define ARRAY_SIZE (ITERS*TOTAL_THREADS)
// GPU error check
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true){
        if (code != hipSuccess) {
                fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
                if (abort) exit(code);
        }
}

//Measure latency of 32768 reads. 
__global__ void l1_lat(uint32_t *startClk, uint32_t *stopClk, float *posArray, float *dsink){	
	// thread index
	uint32_t tid = threadIdx.x;
	uint32_t bid = blockIdx.x;
	uint32_t id = bid * blockDim.x + tid;
	if(tid < blockDim.x && bid < gridDim.x){
		// a register to avoid compiler optimization
		float sink = 0;
		float *ptr = posArray + id;
		// populate l1 cache to warm up
		asm volatile ("{\t\n"
			".reg .f32 data;\n\t"
			"ld.global.cg.f32 data, [%1];\n\t"
			"add.f32 %0, data, %0;\n\t"
			"}" : "+f"(sink) : "l"(ptr) : "memory"
		);
		
		// synchronize all threads
		asm volatile ("bar.sync 0;");
		// start timing
		uint32_t start = 0;
		asm volatile ("mov.u32 %0, %%clock;" : "=r"(start) :: "memory");
		for(uint32_t i=0; i<ITERS; i++) {	
			// load data from l1 cache and accumulate
			asm volatile ("{\t\n"
				".reg .f32 data;\n\t"
				"ld.global.cg.f32 data, [%1];\n\t"
				"add.f32 %0, data, %0;\n\t"
				"}" : "+f"(sink) : "l"(ptr) : "memory"
			);
			// synchronize all threads
			asm volatile("bar.sync 0;");
		}
		// stop timing
		uint32_t stop = 0;
		asm volatile("mov.u32 %0, %%clock;" : "=r"(stop) :: "memory");
		// write time and data back to memory
		startClk[id] = start;
		stopClk[id] = stop;
		dsink[id] = sink;
	}
}

int main(){
	uint32_t *startClk = (uint32_t*) malloc(TOTAL_THREADS*sizeof(uint32_t));
	uint32_t *stopClk = (uint32_t*) malloc(TOTAL_THREADS*sizeof(uint32_t));
	float *posArray = (float*) malloc(TOTAL_THREADS*sizeof(float));
	float *dsink = (float*) malloc(TOTAL_THREADS*sizeof(float));
	
	uint32_t *startClk_g;
        uint32_t *stopClk_g;
        float *posArray_g;
        float *dsink_g;
	
	for (uint32_t i=0; i<TOTAL_THREADS; i++)
		posArray[i] = (float)i;
		
	gpuErrchk( hipMalloc(&startClk_g, TOTAL_THREADS*sizeof(uint32_t)) );
	gpuErrchk( hipMalloc(&stopClk_g, TOTAL_THREADS*sizeof(uint32_t)) );
	gpuErrchk( hipMalloc(&posArray_g, TOTAL_THREADS*sizeof(float)) );
	gpuErrchk( hipMalloc(&dsink_g, TOTAL_THREADS*sizeof(float)) );
	
	gpuErrchk( hipMemcpy(posArray_g, posArray, TOTAL_THREADS*sizeof(float), hipMemcpyHostToDevice) );

	l1_lat<<<BLOCKS_NUM,THREADS_NUM>>>(startClk_g, stopClk_g, posArray_g, dsink_g);

	gpuErrchk( hipMemcpy(startClk, startClk_g, TOTAL_THREADS*sizeof(uint32_t), hipMemcpyDeviceToHost) );
	gpuErrchk( hipMemcpy(stopClk, stopClk_g, TOTAL_THREADS*sizeof(uint32_t), hipMemcpyDeviceToHost) );
	gpuErrchk( hipMemcpy(dsink, dsink_g, TOTAL_THREADS*sizeof(float), hipMemcpyDeviceToHost) );
	printf("L1 Latency for %d threads = %u \n", TOTAL_THREADS, (stopClk[0]-startClk[0]));

	return 0;
} 
