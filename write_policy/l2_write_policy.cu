#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdio>
using namespace std;
#include <hip/hip_runtime.h>
#define TIMES 24

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////HELP FUNCTIONS/////////////////////////////////////////////////
void RandomInit(float* data, int n)
{
    for (int i=0; i<n; i++)
	{
        data[i] = rand() / (float)RAND_MAX;
	}
}

void RandomInit(unsigned* data, int n)
{
    for (int i=0; i<n; i++)
	{
        data[i] = rand() % n;
	}
}
#define checkCudaErrors(err)  __checkCudaErrors (err, __FILE__, __LINE__)
inline void __checkCudaErrors(hipError_t err, const char *file, const int line )
{
    if(hipSuccess != err)
    {
        fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n",file, line, (int)err, hipGetErrorString( err ) );
        exit(-1);        
    }
}

// This will output the proper error string when calling hipGetLastError
#define getLastCudaError(msg)      __getLastCudaError (msg, __FILE__, __LINE__)

inline void __getLastCudaError(const char *errorMessage, const char *file, const int line )
{
    hipError_t err = hipGetLastError();
    if (hipSuccess != err)
    {
        fprintf(stderr, "%s(%i) : getLastCudaError() CUDA error : %s : (%d) %s.\n",
        file, line, errorMessage, (int)err, hipGetErrorString( err ) );
        exit(-1);
    }
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////_VECTOR_ADDITION_///////////////////////////////////////////////////////
// Device code


__global__ void irreguler(const float* A, float* C, float* F)	
	
{	
int i = blockDim.x * blockIdx.x + threadIdx.x;	
	if(i == 0)
	{
	C[i] = A[i];
	__syncthreads();
	//high latncy
	C[i] = C[i] + A[i];
	__syncthreads();
	
	F[i] = C[i+1];
	}
	
}

__global__ void mb2(float* A, float* C)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i == 0) {
	C[i] = A[i]; //write to C[i] is a miss (cache line is missing)
	__syncthreads();
	C[i+1] = A[i]; //write to C[i+1] is a hit (cache line is found)
	__syncthreads();
	C[i] = C[i] + A[i]; //read of C[i] is a miss (entire sector is missing, fetch it from memory)
	__syncthreads();
	A[i] = C[i] + C[i+1]; //read C[i] and C[i+1] are hits (entire sector exists)
	}
}	


__global__ void l1(float* A, float* C)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i == 0) {
		C[i] = A[i] + A[i+32];

                __syncthreads();

        C[i] = A[i] + A[i+32] + C[i];
	}
}	


// Host code
void VectorAddition(int N, int threadsPerBlock)
{
	cout<<"Vector Addition for input size "<<N<<" :\n";
	// Variables
	float* h_A;
	float* h_C;

	
	float* d_A;
	float* d_C;
	
	float total_time=0;
    size_t size = N * sizeof(float);

    // Allocate input vectors h_A and h_B in host memory
    h_A = (float*)malloc(size);
    h_C = (float*)malloc(size);

    
    // Initialize input vectors
    RandomInit(h_A, N);

	
    // Allocate vectors in device memory
    checkCudaErrors( hipMalloc((void**)&d_A, size) );
    checkCudaErrors( hipMalloc((void**)&d_C, size) );

	
    // Copy vectors from host memory to device memory
    checkCudaErrors( hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice) );
	
	checkCudaErrors(hipDeviceSynchronize());
    // Invoke kernel
	cout<<"Invoke Kernel\n";	
	//int threads = 128;
    int blocksPerGrid = ((N+ threadsPerBlock-1) / threadsPerBlock);
  
  
	for (int i = 0; i < 1; i++) {
    l1<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_C);
    getLastCudaError("kernel launch failure");
	checkCudaErrors(hipDeviceSynchronize());
	}

	float dSeconds = total_time/((float)TIMES * 1000);
	float dNumOps = N;
	float gflops = 1.0e-9 * dNumOps/dSeconds;
	cout<<"Time = "<<dSeconds*1.0e3<< "msec"<<endl<<"gflops = "<<gflops<<endl;

    // Copy result from device memory to host memory
    // h_C contains the result in host memory
    checkCudaErrors( hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost) );
    
    // Verify result
    int i;
    for (i = 0; i < N; ++i) {
        float sum = h_A[i];
        if (fabs(h_C[i] - sum) > 1e-5)
            break;
    }

        // Free device memory
    if (d_A)
        hipFree(d_A);
    if (d_C)
        hipFree(d_C);

    // Free host memory
    if (h_A)
        free(h_A);
    if (h_C)
        free(h_C);
        
    hipDeviceReset();

	if(i == N)
		cout<<"SUCCSESS"<<endl;
	else 
		cout<<"FAILED"<<endl;   
}
//////////////////////////////////////////////////////
int main(int argc,char *argv[])
{ 
  if(argc < 3)
     printf("Unsuffcient number of arguments!\n");
else
	{
		VectorAddition(atoi(argv[1]), atoi(argv[2]));
	}
}
